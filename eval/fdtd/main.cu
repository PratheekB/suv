#include "hip/hip_runtime.h"
/**
 * fdtd2d.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "penguin.h"
#include "polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 10.05

#define GPU_DEVICE 0

#define MiB 19252
#define RESERVATION ((MiB*1024UL*1024ULL))

/* Problem size */
#define tmax 10 
#define NX 2048 * 12
#define NY 2048 * 12

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;

bool do_saby = 0;

void init_arrays(DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz, DATA_TYPE* _fict_gpu, DATA_TYPE* ex_gpu, DATA_TYPE* ey_gpu, DATA_TYPE* hz_gpu)
{
	int i, j;

  	for (i = 0; i < tmax; i++)
	{
		_fict_[i] = (DATA_TYPE) i;
		_fict_gpu[i] = (DATA_TYPE) i;
	}
	
	for (i = 0; i < NX; i++)
	{
		for (j = 0; j < NY; j++)
		{
			ex[i*NY + j] = ((DATA_TYPE) i*(j+1) + 1) / NX;
			ey[i*NY + j] = ((DATA_TYPE) (i-1)*(j+2) + 2) / NX;
			hz[i*NY + j] = ((DATA_TYPE) (i-9)*(j+4) + 3) / NX;
			ex_gpu[i*NY + j] = ((DATA_TYPE) i*(j+1) + 1) / NX;
			ey_gpu[i*NY + j] = ((DATA_TYPE) (i-1)*(j+2) + 2) / NX;
			hz_gpu[i*NY + j] = ((DATA_TYPE) (i-9)*(j+4) + 3) / NX;
		}
	}
}


void runFdtd(DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz)
{
	int t, i, j;
	
	for (t=0; t < tmax; t++)  
	{
		for (j=0; j < NY; j++)
		{
			ey[0*NY + j] = _fict_[t];
		}
	
		for (i = 1; i < NX; i++)
		{
       		for (j = 0; j < NY; j++)
			{
       			ey[i*NY + j] = ey[i*NY + j] - 0.5*(hz[i*NY + j] - hz[(i-1)*NY + j]);
        		}
		}

		for (i = 0; i < NX; i++)
		{
       		for (j = 1; j < NY; j++)
			{
				ex[i*(NY+1) + j] = ex[i*(NY+1) + j] - 0.5*(hz[i*NY + j] - hz[i*NY + (j-1)]);
			}
		}

		for (i = 0; i < NX; i++)
		{
			for (j = 0; j < NY; j++)
			{
				hz[i*NY + j] = hz[i*NY + j] - 0.7*(ex[i*(NY+1) + (j+1)] - ex[i*(NY+1) + j] + ey[(i+1)*NY + j] - ey[i*NY + j]);
			}
		}
	}
}


void compareResults(DATA_TYPE* hz1, DATA_TYPE* hz2)
{
	int i, j, fail;
	fail = 0;
	
	for (i=0; i < NX; i++) 
	{
		for (j=0; j < NY; j++) 
		{
			if (percentDiff(hz1[i*NY + j], hz2[i*NY + j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}



__global__ void fdtd_step1_kernel(DATA_TYPE* _fict_, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NX) && (j < NY))
	{
		if (i == 0) 
		{
			ey[i * NY + j] = _fict_[t];
		}
		else
		{ 
			ey[i * NY + j] = ey[i * NY + j] - 0.5f*(hz[i * NY + j] - hz[(i-1) * NY + j]);
		}
	}
}



__global__ void fdtd_step2_kernel(DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < NX) && (j < NY) && (j > 0))
	{
		ex[i * (NY+1) + j] = ex[i * (NY+1) + j] - 0.5f*(hz[i * NY + j] - hz[i * NY + (j-1)]);
	}
}


__global__ void fdtd_step3_kernel(DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < NX) && (j < NY))
	{	
		hz[i * NY + j] = hz[i * NY + j] - 0.7f*(ex[i * (NY+1) + (j+1)] - ex[i * (NY+1) + j] + ey[(i + 1) * NY + j] - ey[i * NY + j]);
	}
}


void fdtdCuda(DATA_TYPE* _fict_gpu, DATA_TYPE* ex_gpu, DATA_TYPE* ey_gpu, DATA_TYPE* hz_gpu)
{
	double t_start, t_end;

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid( (size_t)ceil(((float)NY) / ((float)block.x)), (size_t)ceil(((float)NX) / ((float)block.y)));

	t_start = rtclock();
  nvml_start();

  unsigned long long available = 23860 - MiB - 10;
  available = available * 1024 * 1024; 
  /* if(do_saby) { */
  /*   if(available > sizeof(DATA_TYPE) * NX * (NY+1)) { */
  /*     hipMemPrefetchAsync(ex_gpu, sizeof(DATA_TYPE) * NX * (NY+1), GPU_DEVICE, 0 ); */
  /*     available -= sizeof(DATA_TYPE) * NX * (NY+1); */
  /*   } else { */
  /*     hipMemPrefetchAsync(ex_gpu, available, GPU_DEVICE, 0 ); */
  /*     available = 0; */
  /*   } */
  /*   if(available > sizeof(DATA_TYPE) * NY * (NX+1)) { */
  /*     hipMemPrefetchAsync(ey_gpu, sizeof(DATA_TYPE) * NY * (NX+1), GPU_DEVICE, 0 ); */
  /*     available -= sizeof(DATA_TYPE) * NY * (NX+1); */
  /*   } else { */
  /*     hipMemPrefetchAsync(ey_gpu, available, GPU_DEVICE, 0 ); */
  /*     available = 0; */
  /*   } */
  /*   if(available > sizeof(DATA_TYPE) * NY * (NX)) { */
  /*     hipMemPrefetchAsync(hz_gpu, sizeof(DATA_TYPE) * NY * (NX), GPU_DEVICE, 0 ); */
  /*     available -= sizeof(DATA_TYPE) * NY * (NX); */
  /*   } else { */
  /*     hipMemPrefetchAsync(hz_gpu, available, GPU_DEVICE, 0 ); */
  /*     available = 0; */
  /*   } */
  /*   hipMemAdvise(ex_gpu, sizeof(DATA_TYPE) * NX*(NY+1), hipMemAdviseSetAccessedBy, 0); */
  /*   hipMemAdvise(ey_gpu, sizeof(DATA_TYPE) * (NX+1)*(NY), hipMemAdviseSetAccessedBy, 0); */
  /*   hipMemAdvise(hz_gpu, sizeof(DATA_TYPE) * (NX)*(NY), hipMemAdviseSetAccessedBy, 0); */
  /* } */
  penguinStartStatCollection();
  for(int t = 0; t< tmax; t++)
  {
    fdtd_step1_kernel<<<grid,block>>>(_fict_gpu, ex_gpu, ey_gpu, hz_gpu, t);
    hipDeviceSynchronize();
    fdtd_step2_kernel<<<grid,block>>>(ex_gpu, ey_gpu, hz_gpu, t);
    hipDeviceSynchronize();
    fdtd_step3_kernel<<<grid,block>>>(ex_gpu, ey_gpu, hz_gpu, t);
    hipDeviceSynchronize();
    printf("%d\n", t);
  }
  penguinStopStatCollection();

	t_end = rtclock();
  nvml_stop();
    	fprintf(stdout, "GPU.Parser.Time: %0.6lf\n", t_end - t_start);
}


int main()
{
  int* reservation;
  hipMalloc((void**) &reservation, RESERVATION);

	double t_start, t_end;

	DATA_TYPE* _fict_;
	DATA_TYPE* ex;
	DATA_TYPE* ey;
	DATA_TYPE* hz;

	DATA_TYPE *_fict_gpu;
	DATA_TYPE *ex_gpu;
	DATA_TYPE *ey_gpu;
	DATA_TYPE *hz_gpu;

	_fict_ = (DATA_TYPE*)malloc(tmax*sizeof(DATA_TYPE));
	ex = (DATA_TYPE*)malloc(NX*(NY+1)*sizeof(DATA_TYPE));
	ey = (DATA_TYPE*)malloc((NX+1)*NY*sizeof(DATA_TYPE));
	hz = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));

	hipMallocManaged(&_fict_gpu, sizeof(DATA_TYPE) * tmax);
	hipMallocManaged(&ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1));
	hipMallocManaged(&ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY);
	hipMallocManaged(&hz_gpu, sizeof(DATA_TYPE) * NX * NY);

	init_arrays(_fict_, ex, ey, hz, _fict_gpu, ex_gpu, ey_gpu, hz_gpu);

  /* penguinSetPrioritizedLocation(hz_gpu, sizeof(DATA_TYPE) * (NX+1) * NY, 0); */

	GPU_argv_init();
	fdtdCuda(_fict_gpu, ex_gpu, ey_gpu, hz_gpu);

	t_start = rtclock();
	/* runFdtd(_fict_, ex, ey, hz); */
	t_end = rtclock();
	
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
	
	compareResults(hz, hz_gpu);

	free(_fict_);
	free(ex);
	free(ey);
	free(hz);
			
	hipFree(_fict_gpu);
	hipFree(ex_gpu);
	hipFree(ey_gpu);
	hipFree(hz_gpu);
	return 0;
}

