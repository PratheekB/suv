#include "hip/hip_runtime.h"
/*********************************************************************************/
//
// Polybench kernels implementation on CUDA GPU
//
// Computer & Information Science, University of Delaware
// Author(s):   Sudhee Ayalasomayajula (sudhee1@gmail.com)
//              John Cavazos (cavazos@cis.udel.edu)
//		Scott Grauer Gray(sgrauerg@gmail.com)
//              Robert Searles (rsearles35@aol.com)   
//              Lifan Xu (xulifan@udel.edu)
//
// Contact(s): Lifan Xu (xulifan@udel.edu)
// Reference(s):
//
/*********************************************************************************/

#define MiB 18399
#define RESERVATION ((MiB*1024UL*1024))
#define SABY 0

#include "penguin.h"

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "polybenchUtilFuncts.h"

/* #include "../../common/polybenchUtilFuncts.h" */
/* #include "../../common/polybench.h" */
/* #include "../../common/polybench.c" */

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

/* Problem size. */
#define NR 1024ULL
#define NQ 1024ULL
#define NP 1024ULL

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

#define GPU_DEVICE 0

int do_saby = 0;

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void doitgenCPU(DATA_TYPE *sum, DATA_TYPE *A, DATA_TYPE *C4)
{
	for (int r = 0; r < NR; r++)
	{
		for (int q = 0; q < NQ; q++)  
		{
			for (int p = 0; p < NP; p++)  
			{
				sum[r * (NQ * NP) + q * NP + p] = (DATA_TYPE)0.0;
				for (int s = 0; s < NP; s++)
				{
					sum[r * (NQ * NP) + q * NP + p] = sum[r * (NQ * NP) + q * NP + p] + A[r * (NQ * NP) + q * NP + s] * C4[s * NP + p];
				}
      		}
      		
			for (int p = 0; p < NP; p++)
       		{
				A[r * (NQ * NP) + q * NP + p] = sum[r * (NQ * NP) + q * NP + p];
			}
		}
	}
}


void init_array(DATA_TYPE *A, DATA_TYPE *C4, DATA_TYPE *sum)
{
  	for (int i = 0; i < NR; i++)
  	{
    		for (int j = 0; j < NQ; j++)
    		{
      			for (int k = 0; k < NP; k++)
      			{
	 			A[i * (NQ * NP) + j * NP + k] = ((DATA_TYPE) i*j + k) / NP;
        sum[i * (NQ * NP) + j * NP + k] = (DATA_TYPE) 0.0;
      			}
    		}
  	}

  	for (int i = 0; i < NP; i++)
  	{
    		for (int j = 0; j < NP; j++)
    		{
      			C4[i * NP + j] = ((DATA_TYPE) i*j) / NP;
    		}
  	}
}


void compareResults(DATA_TYPE* sum, DATA_TYPE* sum_outputFromGpu)
{
	int fail = 0;
	
	for (int r = 0; r < NR; r++)
	{
    		for (int q = 0; q < NQ; q++)  
		{
      			for (int p = 0; p < NP; p++)  
			{
				if (percentDiff(sum[r * (NQ * NP) + q * NP + p], sum_outputFromGpu[r * (NQ * NP) + q * NP + p]) > PERCENT_DIFF_ERROR_THRESHOLD)
				{
					fail++;
				}
			}
		}
	}
	
	// Print results
	printf("Number of misses: %d\n", fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void doitgen_kernel1(DATA_TYPE *sum, DATA_TYPE *A, DATA_TYPE *C4, int r)
{
	int p = blockIdx.x * blockDim.x + threadIdx.x;
	int q = blockIdx.y * blockDim.y + threadIdx.y;

	if ((p < NP) && (q < NQ))
	{

    // changes by Pratheek, reverted now
		sum[r * (NQ * NP) + q * NP + p] = (DATA_TYPE)0.0;
		/* DATA_TYPE temp = (DATA_TYPE)0.0; */
	
		for (int s = 0; s < NP; s++)
		{
		sum[r * (NQ * NP) + q * NP + p] = sum[r * (NQ * NP) + q * NP + p] + A[r * (NQ * NP) + q * NP + s] * C4[s * NP + p];
		/* temp = temp + A[r * (NQ * NP) + q * NP + s] * C4[s * NP + p]; */
		}
		/* sum[r * (NQ * NP) + q * NP + p] = temp; */
	}
}

__global__ void doitgen_kernel2(DATA_TYPE *sum, DATA_TYPE *A, DATA_TYPE *C4, int r)
{
	int p = blockIdx.x * blockDim.x + threadIdx.x;
	int q = blockIdx.y * blockDim.y + threadIdx.y;

	if ((p < NP) && (q < NQ))
	{
		A[r * (NQ * NP) + q * NP + p] = sum[r * (NQ * NP) + q * NP + p];
	}
}

void doitgenCuda(DATA_TYPE* A, DATA_TYPE* C4, DATA_TYPE* sum)
{
	double t_start, t_end;

	DATA_TYPE* AGpu = A;
	DATA_TYPE* C4Gpu = C4;
	DATA_TYPE* sumGpu = sum;

	/* hipMallocManaged(&AGpu, NR * NQ * NP * sizeof(DATA_TYPE)); */
	/* hipMallocManaged(&C4Gpu, NP * NP * sizeof(DATA_TYPE)); */
	/* hipMallocManaged(&sumGpu, NR * NQ * NP * sizeof(DATA_TYPE)); */

	/* hipMemcpy(AGpu, A, NR * NQ * NP * sizeof(DATA_TYPE), hipMemcpyHostToDevice); */
	/* hipMemcpy(C4Gpu, C4, NP * NP * sizeof(DATA_TYPE), hipMemcpyHostToDevice); */
	/* hipMemcpy(sumGpu, sum, NR * NQ * NP * sizeof(DATA_TYPE), hipMemcpyHostToDevice); */

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((unsigned int)ceil( ((float)NP) / ((float)block.x) ), (unsigned int)ceil( ((float)NR) / ((float)block.y) ));
	t_start = rtclock();
	
#if SABY
  hipMemAdvise(A, sizeof(DATA_TYPE) * NR * NQ * NP, hipMemAdviseSetAccessedBy, 0);
  hipMemAdvise(C4, sizeof(DATA_TYPE) * NQ * NP, hipMemAdviseSetAccessedBy, 0);
  hipMemAdvise(sum, sizeof(DATA_TYPE) * NR * NQ * NP, hipMemAdviseSetAccessedBy, 0);
#endif

  hipStream_t stream1;
  hipStreamCreate(&stream1);

  penguinSetPrioritizedLocation(C4, sizeof(DATA_TYPE) * NP * NP, 0);
  /* penguinSuperPrefetch(AGpu, 1, 0, 0); */
    /* hipMemPrefetchAsync(AGpu + (0/512)*NR/4*NP*NQ,sizeof(DATA_TYPE)*NR/4*NP*NQ, GPU_DEVICE, 0 ); */
    /* hipMemPrefetchAsync(AGpu, sizeof(DATA_TYPE)*NR*NP*NQ, GPU_DEVICE, 0 ); */

  /* if(do_saby) { */
  /* hipMemAdvise(A, sizeof(DATA_TYPE) * NR * NQ * NP, hipMemAdviseSetAccessedBy, 0); */
  /* hipMemAdvise(C4, sizeof(DATA_TYPE) * NQ * NP, hipMemAdviseSetAccessedBy, 0); */
  /* hipMemAdvise(sum, sizeof(DATA_TYPE) * NR * NQ * NP, hipMemAdviseSetAccessedBy, 0); */
  /*   unsigned MB = 23860-MiB-10; */
  /*   unsigned long long available = (unsigned long long)(MB) * 1024ULL * 1024ULL; */
  /*   printf("av = %llu\n", available); */
  /*   if(available >= (sizeof(DATA_TYPE) * NQ * NP)) { */
  /*     hipMemPrefetchAsync(C4 ,sizeof(DATA_TYPE)*(NQ)*NP, GPU_DEVICE,  0); */
  /*     available -= ((sizeof(DATA_TYPE) * NQ * NP)); */
  /*   } else { */
  /*     hipMemPrefetchAsync(C4 ,available, GPU_DEVICE,  0); */
  /*     available = 0; */
  /*   } */
  /*   printf("av = %llu\n", available); */
  /*   if(available >= (sizeof(DATA_TYPE) * NR * NQ * NP)) { */
  /*     hipMemPrefetchAsync(A, sizeof(DATA_TYPE) * NR * NQ * NP, GPU_DEVICE,  0); */
  /*     available -= ((sizeof(DATA_TYPE) * NR * NQ * NP)); */
  /*   } else { */
  /*     hipMemPrefetchAsync(A, available, GPU_DEVICE,  0); */
  /*     available = 0; */
  /*   } */
  /*   printf("av = %llu\n", available); */
  /*   if(available >= (sizeof(DATA_TYPE) * NR * NQ * NP)) { */
  /*     hipMemPrefetchAsync(sum ,sizeof(DATA_TYPE)*NR * NQ * NP, GPU_DEVICE,  0); */
  /*     available -= ((sizeof(DATA_TYPE) * NR * NQ * NP)); */
  /*   } else { */
  /*     hipMemPrefetchAsync(sum ,available, GPU_DEVICE,  0); */
  /*     available = 0; */
  /*   } */
  /*   printf("av = %llu\n", available); */
  /* } */


  for (int r = 0; r < NR; r++)
  {

    doitgen_kernel1 <<<grid, block>>> (sumGpu, AGpu, C4Gpu, r);
    hipDeviceSynchronize();
    doitgen_kernel2 <<<grid, block>>> (sumGpu, AGpu, C4Gpu, r);
    hipDeviceSynchronize();
  }

	t_end = rtclock();
	fprintf(stdout, "GPU.Parser.Time: %0.6lf\n", t_end - t_start);
	
	/* hipMemcpy(sum_outputFromGpu, sumGpu, NR * NQ * NP * sizeof(DATA_TYPE), hipMemcpyDeviceToHost); */

	hipFree(AGpu);
	hipFree(C4Gpu);
	hipFree(sumGpu);
}
	

int main(int argc, char *argv[])
{
  int* reservation;
  hipMalloc((void**) &reservation, RESERVATION);
	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* C4;
	DATA_TYPE* sum, *sum_outputFromGpu;


	/* A = (DATA_TYPE*)malloc(NR * NQ * NP * sizeof(DATA_TYPE)); */
	/* C4 = (DATA_TYPE*)malloc(NP * NP * sizeof(DATA_TYPE)); */
	/* sum = (DATA_TYPE*)malloc(NR * NQ * NP * sizeof(DATA_TYPE)); */
	/* sum_outputFromGpu = (DATA_TYPE*)malloc(NR * NQ * NP * sizeof(DATA_TYPE)); */

	hipMallocManaged(&A, NR * NQ * NP * sizeof(DATA_TYPE));
	hipMallocManaged(&C4, NP * NP * sizeof(DATA_TYPE));
	hipMallocManaged(&sum, NR * NQ * NP * sizeof(DATA_TYPE));

  /* penguinSetPrioritizedLocation(B_gpu, sizeof(DATA_TYPE) * NK * NJ, 0); */

	init_array(A, C4, sum);

  /* hipMemAdvise(A, sizeof(DATA_TYPE) * NR * NQ * NP, hipMemAdviseSetAccessedBy, 0); */
  /* hipMemAdvise(C4, sizeof(DATA_TYPE) * NQ * NP, hipMemAdviseSetAccessedBy, 0); */
  /* hipMemAdvise(sum, sizeof(DATA_TYPE) * NR * NQ * NP, hipMemAdviseSetAccessedBy, 0); */

  penguinStartStatCollection();
  nvml_start();
	doitgenCuda(A, C4, sum);
  nvml_stop();
  penguinStopStatCollection();

	/* t_start = rtclock(); */
	/* doitgenCPU(sum, A, C4); */
	/* t_end = rtclock(); */

	/* fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start); */
	
	/* compareResults(sum, sum_outputFromGpu); */

	/* free(A); */
	/* free(C4); */
	/* free(sum); */
	/* free(sum_outputFromGpu); */
	
    return 0;
}

